#include "hip/hip_runtime.h"

#include <iostream>
#include <stdlib.h>

__global__ void simple_vec_add(float * inA,
                               float * inB,
                               float * outC,
                               int n)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    if(idx<n)
    {
        outC[idx]=inA[idx]+inB[idx];
    }
}

void fillRandomly(float * v,int n)
{
    for(int i=0;i<n;i++)
    {
        v[i]=(float)rand()/(float)RAND_MAX;
    }
}

bool checkResults(float * A,
                  float * B,
                  float * C,
                  int n)
{
    bool res = true;
    float v;

    int i=0;
    while((res==true) && (i<n))
    {
        v = A[i]+B[i];
        if(C[i]!=v)
        {
            res = false;
        }
        i++;
    }

    return res;
}

int main(int argc,char **argv)
{
    float * hA, * hB, * hC;
    float * dA, * dB, * dC;
    int nElements = 10000;

    // Allocate host memory
    hA = (float*) malloc(nElements*sizeof(float));
    hB = (float*) malloc(nElements*sizeof(float));
    hC = (float*) malloc(nElements*sizeof(float));

    // Fill the input A and B vectors with random data
    fillRandomly(hA,nElements);
    fillRandomly(hB,nElements);

    // Allocate device memory
    hipMalloc((void**)&dA,nElements*sizeof(float));
    hipMalloc((void**)&dB,nElements*sizeof(float));
    hipMalloc((void**)&dC,nElements*sizeof(float));

    // Transfer data from host to device
    hipMemcpy(dA,hA,nElements*sizeof(float),hipMemcpyHostToDevice);    
    hipMemcpy(dB,hB,nElements*sizeof(float),hipMemcpyHostToDevice);

    // Perform CUDA kernel computation
    int nThreadsPerBlock = 256;
    dim3 dimGrid((nElements-1)/nThreadsPerBlock+1,1,1);
    dim3 dimBlock(nThreadsPerBlock,1,1);
    simple_vec_add<<<dimGrid,dimBlock>>>(dA,dB,dC,nElements);
    hipDeviceSynchronize();

    // Transfer data from device to host
    hipMemcpy(hC,dC,nElements*sizeof(float),hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    // Check results
    bool ok = checkResults(hA,hB,hC,nElements);
    if(ok)
    {
        std::cout<<"OK"<<std::endl;
    }
    else
    {
        std::cout<<"FAIL"<<std::endl;    
    }

    // Deallocate host memory
    free(hA);
    free(hB);
    free(hC); 

    return 0;
}
